#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    float *d_A;
    size_t size = 50000 * sizeof(float);

    hipError_t err = hipMalloc((void**)&d_A, size);
    if (err != hipSuccess) {
        std::cerr << "Error allocating memory on GPU: " << err << "---" << hipGetErrorString(err) << std::endl;
        return -1;
    }

    std::cout << "Memory allocated successfully!" << std::endl;
    hipFree(d_A);
    return 0;
}
